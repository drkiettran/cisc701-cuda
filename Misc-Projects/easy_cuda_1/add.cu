#include "hip/hip_runtime.h"
﻿#include <assert.h>
#include <stdint.h>
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float* x, float* y)
{
    for (int i = 0; i < n; i++)
        y[i] = x[i] + y[i];
}

int simple_add(int N)
{
    // int N = 1 << 20;
    float* x, * y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the GPU
    add << <1, 1 >> > (N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}

// Kernel function to add the elements of two arrays
__global__
void block_add(int n, float* x, float* y)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int block_add(int N)
{
    // int N = 1 << 20;
    float* x, * y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the GPU
    block_add << <1, 256 >> > (N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}

// Kernel function to add the elements of two arrays
__global__
void grid_add(int n, float* x, float* y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int grid_add(int N)
{
    // int N = 1 << 30;
    float* x, * y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run kernel on 1M elements on the GPU
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    std::cout << "Number of blocks: " << numBlocks << std::endl;
    grid_add << <numBlocks, blockSize >> > (N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}

int main(int argc, const char** argv) {
    // If the command-line has a device number specified, use it
    char* app_name = NULL;
    int N = 1 << 20;

    if (checkCmdLineFlag(argc, argv, "e")) {
        N = 1 << getCmdLineArgumentInt(argc, argv, "e");
    }
    std::cout << "N: " << N << std::endl;
    if (checkCmdLineFlag(argc, argv, "app")) {
        getCmdLineArgumentString(argc, argv, "app", &app_name);
        std::cout << "app=" << app_name << std::endl;

        if (_strnicmp(app_name, "simple_add", strlen(app_name)) == 0) {
            simple_add(N);
        } else if (STRNCASECMP(app_name, "block_add", strlen(app_name)) == 0) {
            block_add(N);
        } else if (STRNCASECMP(app_name, "grid_add", strlen(app_name)) == 0) {
            grid_add(N);
        } else {
            std::cout << "Invalid app name: " << app_name << std::endl;
            exit(EXIT_FAILURE);
        }
    }
    else {
        std::cout << "Must provide -app= parameters!" << std::endl;
    }
}